#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <ctime>

#define BLOCK_SIZE 32

using namespace std;

#define CHECK_ERROR(call) do {                                         
   if( hipSuccess != call) {                                          
      std::cerr << std::endl << "CUDA ERRO: " <<                       
         hipGetErrorString(call) <<  " in file: " << __FILE__         
         << " in line: " << __LINE__ << std::endl;                      
         exit(0);                                                      
	}	} while (0)

__global__ void MyKernel(float* output, const float value){

}


//Primeiro método a ser utilizado de forma sequencial
float* method1(int time, float* t, float* t1){
	while(time){
		for(int i=1; i<TAM-1; i++){
			for(int j=1; j<TAM-1; j++){
				t1[(i*TAM)+j] = 0.25 * ( t[((i-1)*TAM)+j] +
				t[((i+1)*TAM)+j] +
				t[(i*TAM)+j-1] +
				t[(i*TAM)+j+1] -
				(4 * t[(i*TAM)+j]) ) 
				+ t[(i*TAM)+j];
			}
		}

		float* swap = t1;
		t1 = t;
		t = swap;
		time--;
	}

	return t;
}


__global__ void method1(float* mat, float* matres){

}


int main(int argc, char const *argv[])
{
	int tam = (int)atoi(argv[1]);

	float* t = build_matrix(); //Matriz com os valores iniciais
	float* t1 = build_matrix(); //Matriz com os valores atualizados

	hipEvent_t start; 
	hipEvent_t stop;  

	double* d_mat1, d_mat2;

	//Alocando espaço de memória
	CHECK_ERROR(hipMalloc((void**)&d_mat1, size_t(TAM*TAM)*sizeof(double)));
	CHECK_ERROR(hipMalloc((void**)&d_mat2, size_t(TAM*TAM)*sizeof(double)));

	method1 <<<1, tam>>>(d_mat1, d_mat2);


	//Desalocando memória
	hipFree(d_mat1);
	hipFree(d_mat2);

	return 0;
}