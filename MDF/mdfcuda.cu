#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <mpi.h>
#include <stdio.h>
#define ITERACOES 200
#define SIZE 1024
#define TAM 512


__global__ void MatrixProblem(float *matriz, float *matrizResultado)
{
    // int tid = threadIdx.x;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid/TAM > 0 && tid/TAM < TAM-1 && tid%TAM != 0 && tid%TAM != TAM-1){
    	matrizResultado[tid] = (0.25*(matriz[tid+1] + matriz[tid-1] + matriz[tid-TAM] + matriz[tid + TAM] - (4*matriz[tid]))) + matriz[tid];
	}
	else{
		matrizResultado[tid] = 100;
	}
}

int main()
{
	float *pMatriz;
	float *pMatrizResultado;
	float mat[TAM][TAM];
	float mCopy[TAM][TAM];
	double t0, t1;

	//preenchendo com 0
	for(int j = 0; j<TAM; j++){
		for(int i = 0; i<TAM; i++){
			mat[i][j] = 0;
			mCopy[i][j] = 0;
		}
	}

	//inicializando valores
	for(int i=0; i<TAM; i++){
		for (int j = 0; j<TAM; j++){
			if(i == 0 || j == 0 || i == TAM-1 || j == TAM-1){
				mat[i][j] = 100;
				mCopy[i][j] = 100;
			}
		}
	}

	hipMalloc((void **)&pMatriz, sizeof(float) * size_t(TAM*TAM)); 
	hipMalloc((void **)&pMatrizResultado, sizeof(float) * size_t(TAM*TAM)); 

	t0 = MPI_Wtime();
	for (int i = 0; i < ITERACOES; ++i){
	    hipMemcpy(pMatriz, mat, sizeof(float) * size_t(TAM*TAM), hipMemcpyHostToDevice); 

		MatrixProblem <<<256, SIZE>>>(pMatriz, pMatrizResultado);

		hipDeviceSynchronize();
		hipMemcpy(mat, pMatrizResultado, sizeof(float) * size_t(TAM*TAM), hipMemcpyDeviceToHost);
	}
		
	t1 = MPI_Wtime();

	printf("O processo terminou em %f segundos\n", t1-t0);
    // cout << "terminou em " << t1-t0 << " segundos" << endl;

	// for(int j = 0; j<TAM; j++){
	// 	for(int i = 0; i<TAM; i++){
	// 		printf("%f ", mat[j][i]);
	// 	}
	// 	printf("\n");
	// }

	hipFree(pMatriz);
	hipFree(pMatrizResultado);

	return 0;
}
