#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <mpi.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define TAM 512
#define TEMP 100
#define MASTER 0
#define TPROCESS 1
#define TIMER 200

using namespace std;

__global__ void MatrixProblem(float *matriz, float *matrizResultado)
{
  // int tid = threadIdx.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if(tid < TAM*TAM/2){
      if(tid/TAM > 0 && tid/TAM < TAM-1 && tid%TAM != 0 && tid%TAM != TAM-1){
        matrizResultado[tid] = (0.25*(matriz[tid+1] + matriz[tid-1] + matriz[tid-TAM] + matriz[tid + TAM] - (4*matriz[tid]))) + matriz[tid];
    }
    else{
      matrizResultado[tid] = 100;
    }
  }
}

__global__ void MatrixProblem2(float *matriz, float *matrizResultado)
{
  // int tid = threadIdx.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if(tid >= TAM*TAM/2 && tid <= TAM*TAM){
      if(tid/TAM > 0 && tid/TAM < TAM-1 && tid%TAM != 0 && tid%TAM != TAM-1){
        matrizResultado[tid] = (0.25*(matriz[tid+1] + matriz[tid-1] + matriz[tid-TAM] + matriz[tid + TAM] - (4*matriz[tid]))) + matriz[tid];
    }
    else{
      matrizResultado[tid] = 100;
    }
  }
}

#define CHECK_ERROR(call) do {                                                    \
   if( hipSuccess != call) {                                                             \
      std::cerr << std::endl << "CUDA ERRO: " <<                             \
         hipGetErrorString(call) <<  " in file: " << __FILE__                \
         << " in line: " << __LINE__ << std::endl;                               \
         exit(0);                                                                                 \
   } } while (0)


float* method1(size_t inicio, size_t fim, float* t, float* t1);
float* method2(size_t inicio, size_t fim, float* t, float* t1);
float* build_matrix();
void print_matrix(float* mat);

void print_matrix(float* mat){
  for(int i=0; i<TAM; i++){
    for(int j=0; j<TAM; j++){
      int k = (i*TAM)+j;
      cout << fixed << setprecision(1) << " " << mat[k];
    }
    cout << endl;
  }
}

float* build_matrix(){
  float* mat = (float*)malloc(TAM * TAM * sizeof(float));

  if(!mat){
    cout << "Espaço para alocação de memória insuficiente" << endl;
    exit(1);
  }

  for(int i=0; i<TAM; i++){
    for(int j=0; j<TAM; j++){
      int k = (i*TAM)+j;
      if(i == 0 || j == 0 || i == TAM-1 || j == TAM-1){
        mat[k] = TEMP;
      } else{
        mat[k] = 0;
      }
    }
  }

  return mat;
}

float* method1(size_t inicio, size_t fim, float* t, float* t1){
    for(int i=inicio; i<fim-1; i++){
      for(int j=1; j<TAM-1; j++){
        t1[(i*TAM)+j] = 0.25 * ( t[((i-1)*TAM)+j] +
        t[((i+1)*TAM)+j] +
        t[(i*TAM)+j-1] +
        t[(i*TAM)+j+1] -
        (4 * t[(i*TAM)+j]) )
        + t[(i*TAM)+j];
      }
    }

  return t1;
}

float* method2(size_t inicio, size_t fim, float* t, float* t1){
  for(int i=inicio; i<fim-1; i++){
    for(int j=1; j<TAM-1; j++){
      t1[(i*TAM)+j] = ( (4 * t[((i-1)*TAM)+j] +
              t[((i+1)*TAM)+j] +
              t[(i*TAM)+j-1] +
              t[(i*TAM)+j+1]) +
              ( t[((i+1)*TAM)+j+1] +
              t[((i+1)*TAM)+j-1] +
              t[((i-1)*TAM)+j-1] +
              t[((i-1)*TAM)+j+1] ) )/20;
     }
   }

  return t1;
}

int main(int argc, char **argv) {
    int id_task, //Número do processo
    num_task, //Threads
    namelen, //Nome da máquina que o processador está sendo usado
    elmt_task, //Quantidade de elementos na comunicação
    tag_task, //Identificador da comunicação
    work_tasks, //Divisão da matriz por threads
    mdftime, //Número de iterações
    timertotal;

    float *pMatriz;
    float *pMatrizResultado;
    float *pMatriz2;
    float *pMatrizResultado2;

    hipStream_t streams[2];
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);

    mdftime = TIMER;
    timertotal = mdftime;
    float* m0 = build_matrix();
    float* m1 = build_matrix();

    double t0, t1; //Tempo de início de fim

    MPI_Status status;

    char processor_name[MPI_MAX_PROCESSOR_NAME];

    MPI_Init(&argc,&argv);
    MPI_Comm_size(MPI_COMM_WORLD,&num_task);
    MPI_Comm_rank(MPI_COMM_WORLD,&id_task);
    MPI_Get_processor_name(processor_name,&namelen);

    work_tasks = TAM/num_task;

    // cout << timertotal << endl;

    hipMalloc((void **)&pMatriz, sizeof(float) * size_t(TAM*TAM)); 
    hipMalloc((void **)&pMatrizResultado, sizeof(float) * size_t(TAM*TAM)); 
    hipMalloc((void **)&pMatriz2, sizeof(float) * size_t(TAM*TAM)); 
    hipMalloc((void **)&pMatrizResultado2, sizeof(float) * size_t(TAM*TAM)); 

    t0 = MPI_Wtime();

    while(mdftime){
      if(id_task == MASTER){
        size_t temp = work_tasks;
        temp *= TAM;

        if(mdftime < TIMER){
          for (size_t i = 1; i < TAM-1; i++) {
            MPI_Recv(&m0[temp+i], 1, MPI_FLOAT, id_task+1, TPROCESS, MPI_COMM_WORLD, &status);
          }
        }

        // print_matrix(m0);
        hipMemcpyAsync(pMatriz, m0, sizeof(float) * size_t(TAM*TAM), hipMemcpyHostToDevice, streams[0]); 

        MatrixProblem <<<256, 1024,0, streams[0]>>>(pMatriz, pMatrizResultado);
        hipDeviceSynchronize();
        hipMemcpyAsync(m0, pMatrizResultado, sizeof(float) * size_t(TAM*TAM), hipMemcpyDeviceToHost, streams[0]);

        // cout << "-------PRINT PROCESSO " << id_task << " ITERAÇÃO " << timertotal - mdftime + 1 << "-------" << endl;
        // print_matrix(m0);

        //Envia a borda para o próximo processo
        temp = work_tasks-1;
        temp *= TAM;
        for (size_t i = 1; i < TAM-1; i++) {
          MPI_Send(&m0[temp+i], 1, MPI_FLOAT, id_task+1, TPROCESS, MPI_COMM_WORLD);
        }

        mdftime--;
      } else{
        //Recebe a borda do processo anterior
        size_t temp = work_tasks-1;
        temp *= TAM;

        for (size_t i = 1; i < TAM-1; i++) {
          if(timertotal - mdftime + 1 > 1){
          MPI_Recv(&m0[temp+i], 1, MPI_FLOAT, id_task-1, TPROCESS, MPI_COMM_WORLD, &status);
            // printf("%f\n", m0[temp+i]);
            // printf("%d\n", timertotal - mdftime + 1);        
          }
        }
    
        hipMemcpyAsync(pMatriz2, m0, sizeof(float) * size_t(TAM*TAM), hipMemcpyHostToDevice, streams[1]); 
        MatrixProblem2 <<<256, 1024, 0, streams[1]>>>(pMatriz2, pMatrizResultado2);
        hipDeviceSynchronize();
        hipMemcpyAsync(m0, pMatrizResultado2, sizeof(float) * size_t(TAM*TAM), hipMemcpyDeviceToHost, streams[1]);

        // cout << "-------PRINT PROCESSO " << id_task << " ITERAÇÃO " << timertotal - mdftime + 1 << "-------" << endl;
        // print_matrix(m0);

        temp = work_tasks;
        temp *= TAM;
        for (size_t i = 1; i < TAM-1; i++) {
          MPI_Send(&m0[temp+i], 1, MPI_FLOAT, id_task-1, TPROCESS, MPI_COMM_WORLD);
        }

        mdftime--;
      }
    }

    t1 = MPI_Wtime();

    cout << "terminou em " << t1-t0 << " segundos" << endl;

    if(id_task == MASTER){
        // print_matrix(m0);
    }
    else{
        // print_matrix(m0);
    }

    free(m0);
    free(m1);
    hipStreamDestroy(streams[0]); 
    hipStreamDestroy(streams[1]); 
    hipFree(pMatriz);
    hipFree(pMatrizResultado);
    hipFree(pMatriz2);
    hipFree(pMatrizResultado2);
    MPI_Finalize();

    return EXIT_SUCCESS;
}